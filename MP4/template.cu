#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "CUDA error: ", hipGetErrorString(err));              \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      return -1;                                                          \
    }                                                                     \
  } while (0)

//@@ Define any useful program-wide constants here
#define TILE_WIDTH 4
#define MASK_WIDTH 3

//@@ Define constant memory for device kernel here
__constant__ float kernel[MASK_WIDTH][MASK_WIDTH][MASK_WIDTH];

__global__ void conv3d(float *input, float *output, const int z_size,
                       const int y_size, const int x_size) {
  //@@ Insert kernel code here
  __shared__ float tile[TILE_WIDTH + MASK_WIDTH - 1][TILE_WIDTH + MASK_WIDTH - 1][TILE_WIDTH + MASK_WIDTH - 1];

  int row_o = blockIdx.y * TILE_WIDTH + threadIdx.y;
  int col_o = blockIdx.x * TILE_WIDTH + threadIdx.x;
  int dep_o = blockIdx.z * TILE_WIDTH + threadIdx.z;

  int row_i = row_o - (MASK_WIDTH / 2);
  int col_i = col_o - (MASK_WIDTH / 2);
  int dep_i = dep_o - (MASK_WIDTH / 2);

  if ((row_i >= 0 && row_i < y_size) && (col_i >= 0 && col_i < x_size) && (dep_i >= 0 && dep_i < z_size)){
    tile[threadIdx.z][threadIdx.y][threadIdx.x] = input[dep_i * (x_size * y_size) + row_i * (x_size) + col_i];
  }
  else{
    tile[threadIdx.z][threadIdx.y][threadIdx.x] = 0.0f;
  }

  __syncthreads();

  float p = 0;
  if (threadIdx.z < TILE_WIDTH && threadIdx.y < TILE_WIDTH && threadIdx.x < TILE_WIDTH){
    for (int i = 0; i < MASK_WIDTH; i++){
      for (int j = 0; j < MASK_WIDTH; j++){
        for (int k = 0; k < MASK_WIDTH; k++){
          p += kernel[i][j][k] * tile[i+threadIdx.z][j+threadIdx.y][k+threadIdx.x];
        }
      }
    }

    if (row_o < y_size && col_o < x_size && dep_o < z_size){
      output[dep_o * (x_size * y_size) + row_o * (x_size) + col_o] = p;
    }
  }

  

}

int main(int argc, char *argv[]) {
  wbArg_t args;
  int z_size;
  int y_size;
  int x_size;
  int inputLength, kernelLength;
  float *hostInput;
  float *hostKernel;
  float *hostOutput;
  float *deviceInput;
  float *deviceOutput;

  args = wbArg_read(argc, argv);

  // Import data
  hostInput = (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
  hostKernel =
      (float *)wbImport(wbArg_getInputFile(args, 1), &kernelLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));

  // First three elements are the input dimensions
  z_size = hostInput[0];
  y_size = hostInput[1];
  x_size = hostInput[2];
  wbLog(TRACE, "The input size is ", z_size, "x", y_size, "x", x_size);
  assert(z_size * y_size * x_size == inputLength - 3);
  assert(kernelLength == 27);

  wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

  wbTime_start(GPU, "Doing GPU memory allocation");
  //@@ Allocate GPU memory here
  // Recall that inputLength is 3 elements longer than the input data
  // because the first  three elements were the dimensions

  hipMalloc(&deviceInput, x_size * y_size * z_size * sizeof(float));
  hipMalloc(&deviceOutput, x_size * y_size * z_size * sizeof(float));
  wbTime_stop(GPU, "Doing GPU memory allocation");

  wbTime_start(Copy, "Copying data to the GPU");
  //@@ Copy input and kernel to GPU here
  // Recall that the first three elements of hostInput are dimensions and
  // do
  // not need to be copied to the gpu

  hipMemcpy(deviceInput, hostInput+3, x_size * y_size * z_size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(kernel), hostKernel, MASK_WIDTH * MASK_WIDTH * MASK_WIDTH * sizeof(float));
  
  wbTime_stop(Copy, "Copying data to the GPU");

  wbTime_start(Compute, "Doing the computation on the GPU");
  //@@ Initialize grid and block dimensions here
  dim3 dimGrid(ceil(1.0 * x_size / TILE_WIDTH), ceil(1.0 * y_size / TILE_WIDTH), ceil(1.0 * z_size / TILE_WIDTH));
  dim3 dimBlock(TILE_WIDTH + MASK_WIDTH - 1, TILE_WIDTH + MASK_WIDTH - 1, TILE_WIDTH + MASK_WIDTH - 1);
  //@@ Launch the GPU kernel here
  conv3d<<<dimGrid, dimBlock>>>(deviceInput, deviceOutput, z_size, y_size, x_size);
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Doing the computation on the GPU");

  wbTime_start(Copy, "Copying data from the GPU");
  //@@ Copy the device memory back to the host here
  // Recall that the first three elements of the output are the dimensions
  // and should not be set here (they are set below)
  hipMemcpy(hostOutput+3, deviceOutput, x_size * y_size * z_size * sizeof(float), hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying data from the GPU");

  wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");
  
  // Set the output dimensions for correctness checking
  hostOutput[0] = z_size;
  hostOutput[1] = y_size;
  hostOutput[2] = x_size;
  wbSolution(args, hostOutput, inputLength);

  // Free device memory
  hipFree(deviceInput);
  hipFree(deviceOutput);

  // Free host memory
  free(hostInput);
  free(hostOutput);
  return 0;
}